#include "hip/hip_runtime.h"
#include "bitnet_kernels.h"

extern "C" void bitlinear_int8xint2(int8_t* input0, int8_t* input1, __hip_bfloat16* output0, __hip_bfloat16* s, __hip_bfloat16* ws, int M, int N, int K, hipStream_t stream){
    if (M == 1 && N == 3840 && K == 2560){
        ladder_int8xint2_kernel<1, 3840, 2560, 3, 8, 16><<<dim3(240, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else if (M == 1 && N == 2560 && K == 2560){
        ladder_int8xint2_kernel<1, 2560, 2560, 1, 8, 16><<<dim3(160, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else if (M == 1 && N == 13824 && K == 2560){
        ladder_int8xint2_kernel<1, 13824, 2560, 2, 8, 16><<<dim3(864, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else if (M == 1 && N == 2560 && K == 6912){
        ladder_int8xint2_kernel<1, 2560, 6912, 1, 8, 16><<<dim3(160, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else if(M == 1 && N == 4800 && K == 3200){
        ladder_int8xint2_kernel<1, 4800, 3200, 6, 8, 16><<<dim3(300, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else if(M == 1 && N == 3200 && K == 3200){
        ladder_int8xint2_kernel<1, 3200, 3200, 1, 8, 16><<<dim3(200, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else if(M == 1 && N == 20480 && K == 3200){
        ladder_int8xint2_kernel<1, 20480, 3200, 2, 8, 16><<<dim3(1280, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else if(M == 1 && N == 3200 && K == 10240){
        ladder_int8xint2_kernel<1, 3200, 10240, 1, 8, 16><<<dim3(200, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }    
    else if(M == 1 && N == 5120 && K == 27648){
        ladder_int8xint2_kernel<1, 5120, 27648, 1, 8, 16><<<dim3(320, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else if(M == 1 && N == 55296 && K == 5120){
        ladder_int8xint2_kernel<1, 55296, 5120, 1, 8, 16><<<dim3(3456, 1, 1), dim3(8, 16, 1), 0, stream>>>(input0, input1, output0, s, ws);
    }
    else{
        std::cout << "required ladder gemm kernel: M " << M << ", N " << N << ", K " << K << std::endl;
    }
}